#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void add1D(int* A, int* B, int* C, int nx, int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    int idx = iy * nx + ix;

    C[idx] = A[idx] + B[idx];
}

__global__ void add2D(int* A, int* B, int* C, int nx, int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
        C[idx] = A[idx] + B[idx];
}

__global__ void add1D1D(int* A, int* B, int* C, int nx, int ny) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (ix < nx) {
        for (int iy = 0; iy < ny; ++iy) {
            int idx = iy * nx + ix;
            C[idx] = A[idx] + B[idx];
        }
    }
}


__global__ void add2D1D(int* A, int* B, int* C, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;

    unsigned int idx = iy*nx + ix;

    if (ix < nx && iy < ny)
        C[idx] = A[idx] + B[idx];
}

void sumh (int *A, int *B, int *C, const int nx, const int ny)
{
    int *ia = A;
    int *ib = B;
    int *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx;
        ib += nx;
        ic += nx;
    }
}

void checkres(int *host, int *gpu, const int N)
{
    const double e = 1.0E-6;

    for (int i = 0; i < N; i++)
    {
        if (abs(host[i] - gpu[i]) > e)
        {
            printf("host ", host[i], " gpu ", gpu[i]);
            printf("Test failed\n\n");
            break;
        }
    }
}

void getrandommatrix(int* m, int n)
{
    for (int i=0; i < n; ++i)
        m[i] = rand()% 10;
}

int main( void ) {
    double time1, time2, time3, time4;

    // size of matrix
    unsigned int nx = 1 << 10; // столбцы
    unsigned int ny = 1 << 10; // строки
    int size = nx * ny;

    int* hA = (int*)malloc(size * sizeof(int));
    int* hB = (int*)malloc(size * sizeof(int));
    int* hC = (int*)malloc(size * sizeof(int));
    int* cpuC = (int*)malloc(size * sizeof(int));

    getrandommatrix(hA, size);
    getrandommatrix(hB, size);
    sumh(hA, hB, cpuC, nx, ny);

    int* dA;
    int* dB;
    hipMalloc((void**)&dA, size * sizeof(int));
    hipMalloc((void**)&dB, size * sizeof(int));

    hipMemcpy(dA, hA, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size * sizeof(int), hipMemcpyHostToDevice);

    printf("Started succesfylly\n");

    // 1D
    int* dC1D;
    hipMalloc((void**)&dC1D, size * sizeof(int));

    hipDeviceSynchronize();
    time1 = cpuSecond();
    add1D<<< ny, nx >>>(dA, dB, dC1D, nx, ny);
    hipDeviceSynchronize();
    time1 = cpuSecond() - time1;
    printf("1D <<<", nx, " ", ny, ">>> elapsed ", time1, " ms\n");

    hipMemcpy(hC, dC1D, size * sizeof(int), hipMemcpyDeviceToHost);
    checkres(cpuC, hC, size);
    hipFree(dC1D);

    // 2D
    int* dC2D;
    hipMalloc((void**)&dC2D, size * sizeof(int));

    int dimx = 32;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    hipDeviceSynchronize();
    time2 = cpuSecond();
    add2D<<<grid, block>>>(dA, dB, dC2D, nx, ny);
    hipDeviceSynchronize();
    time2 = cpuSecond() - time2;
    printf("2D <<<", grid.x, grid.y, ", ", block.x, block.y, ">>> elapsed ", time2, " ms\n");

    hipMemcpy(hC, dC2D, size * sizeof(int), hipMemcpyDeviceToHost);
    checkres(cpuC, hC, size);
    hipFree(dC2D);

    // 1D-сетка, 1D-блоки
    int* dC1D1D;
    hipMalloc((void**)&dC1D1D, size * sizeof(int));

    block = dim3{128,1};
    grid = dim3{(nx+block.x-1)/block.x,1};

    hipDeviceSynchronize();
    time3 = cpuSecond();
    add1D1D <<<grid, block>>> (dA, dB, dC1D1D, nx, ny);
    hipDeviceSynchronize();
    time3 = cpuSecond() - time3;
    printf("1D1D <<<", grid.x, grid.y, ", ", block.x, block.y, ">>> elapsed ", time3, " ms\n");
    hipMemcpy(hC, dC1D1D, size * sizeof(int), hipMemcpyDeviceToHost);
    checkres(cpuC, hC, size);
    hipFree(dC1D1D);


    // 2D-сетка, 1D-блоки
    int* dC2D1D;
    hipMalloc((void**)&dC2D1D, size * sizeof(int));

    block = dim3{256};
    grid = dim3{(nx + block.x - 1) / block.x,ny};

    hipDeviceSynchronize();
    time4 = cpuSecond();
    add2D1D<<<grid, block>>> (dA, dB, dC2D1D, nx, ny);
    hipDeviceSynchronize();
    time4 = cpuSecond() - time4;
    printf("2D1D <<<", grid.x, grid.y, ", ", block.x, block.y, ">>> elapsed ", time3, " ms\n");
    hipMemcpy(hC, dC2D1D, size * sizeof(int), hipMemcpyDeviceToHost);
    checkres(cpuC, hC, size);
    hipFree(dC2D1D);


    hipFree(dA);
    hipFree(dB);

    free(hA);
    free(hB);
    free(hC);

    return 0;
}
